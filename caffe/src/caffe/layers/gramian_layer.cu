#include "hip/hip_runtime.h"
#include "caffe/layers/gramian_layer.hpp"

namespace caffe {

template <typename Dtype>
void GramianLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  for (int batch = 0; batch < bottom[0]->shape(0); ++batch) {
    const Dtype* bottom_data = bottom[0]->gpu_data() + bottom[0]->offset(batch);
    Dtype* mutable_top_data = top[0]->mutable_gpu_data() + top[0]->offset(batch);
    caffe_gpu_gemm<Dtype>(
        CblasNoTrans, 
        CblasTrans, 
        bottom[0]->shape(1),
        bottom[0]->shape(1),
        bottom[0]->count(2),
        normalize_scale_,
        bottom_data,
        bottom_data,
        (Dtype)0.,
        mutable_top_data);
  }
}

template <typename Dtype>
void GramianLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down,
      const vector<Blob<Dtype>*>& bottom) {
  for (int batch = 0; batch < bottom[0]->shape(0); ++batch) {
    Dtype* mutable_bottom_diff = bottom[0]->mutable_gpu_diff() + bottom[0]->offset(batch);
    const Dtype* bottom_data = bottom[0]->gpu_data() + bottom[0]->offset(batch);
    const Dtype* top_diff = top[0]->gpu_diff() + top[0]->offset(batch);
    // Input gradient
    if (propagate_down[0]) {
      caffe_gpu_gemm<Dtype>(
          CblasNoTrans,
          CblasNoTrans,
          bottom[0]->shape(1),
          bottom[0]->count(2),
          bottom[0]->shape(1),
          normalize_scale_,
          top_diff,
          bottom_data,
          (Dtype)0.,
          mutable_bottom_diff);
      caffe_gpu_gemm<Dtype>(
          CblasTrans,
          CblasNoTrans,
          bottom[0]->shape(1),
          bottom[0]->count(2),
          bottom[0]->shape(1),
          normalize_scale_,
          top_diff,
          bottom_data,
          (Dtype)1.,
          mutable_bottom_diff);
      // normalize gradient
      if (this->layer_param_.gramian_param().normalize_output()) {
        Dtype L1 = 1;
        caffe_gpu_asum<Dtype>(bottom[0]->count(1), mutable_bottom_diff, &L1);
        // to prevent divide by zero errors
        L1 += 1e-8;
        // divide by L1 norm
        caffe_gpu_div_scalar<Dtype>(
            bottom[0]->count(1),
            L1,
            mutable_bottom_diff);
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(GramianLayer);

}  // namespace caffe
